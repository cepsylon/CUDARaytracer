#include "hip/hip_runtime.h"
#include "importer.cuh"

#include "debug.h"
#include "vec3.cuh"
#include "sphere.cuh"
#include "scene.cuh"

#include <fstream>
#include <string>

namespace importer
{

__global__ void add_sphere_to_scene(Scene * scene, Material material, vec3 position, float radius)
{
	scene->add(new Sphere{ material, position, radius });
}

__global__ void add_light_to_scene(Scene * scene, vec3 position, vec3 intensity)
{
	scene->add(PointLight{ position, intensity });
}

__global__ void set_scene_camera(Scene * scene, vec3 position, vec3 right, vec3 up, vec3 center)
{
	scene->set_camera(Camera{ position, right, up, center });
}

void import_scene(const char * path, Scene * scene)
{
	std::ifstream file{ path };
	if (file.is_open())
	{
		// Parse
		std::string line;
		while (std::getline(file, line))
		{
			switch (line[0])
			{
			case 'C':
			{
				vec3 position, right, up, center;
				
				// Projection center, right, up, distance to projection center
				sscanf_s(line.c_str(), "C (%f,%f,%f) (%f,%f,%f) (%f,%f,%f) %f",
					&center.x, &center.y, &center.z,
					&right.x, &right.y, &right.z, 
					&up.x, &up.y, &up.z, 
					&position.x);

				// Compute position
				position = center + vec3::normalize(vec3::cross(right, up)) * position.x;

				// Upload to GPU
				set_scene_camera <<<1,1>>>(scene, position, right, up, center);
				CheckCUDAError(hipGetLastError());
				CheckCUDAError(hipDeviceSynchronize());
				break;
			}
			case 'L':
			{
				vec3 position, intensity;

				// Position and intensity
				sscanf_s(line.c_str(), "L (%f,%f,%f) (%f,%f,%f)",
					&position.x, &position.y, &position.z,
					&intensity.x, &intensity.y, &intensity.z);

				add_light_to_scene<<<1,1>>>(scene, position, intensity);
				CheckCUDAError(hipGetLastError());
				CheckCUDAError(hipDeviceSynchronize());
				break;
			}
			case 'S':
			{
				vec3 position, color;
				float radius, specular_coefficient, shininess;

				// Position and radius
				sscanf_s(line.c_str(), "S (%f,%f,%f) %f",
					&position.x, &position.y, &position.z, &radius);

				// Material
				// Color, specular coefficient and shininess
				std::getline(file, line);
				sscanf_s(line.c_str(), "(%f,%f,%f) %f %f", &color.r, &color.g, &color.b, &specular_coefficient, &shininess);

				// Upload to GPU
				add_sphere_to_scene << <1, 1 >> > (scene, Material{ color, specular_coefficient, shininess }, position, radius);
				CheckCUDAError(hipGetLastError());
				CheckCUDAError(hipDeviceSynchronize());
				break;
			}
			default:
				break;
			}
		}
	}
}

}
