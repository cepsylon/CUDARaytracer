#include "hip/hip_runtime.h"
#include "importer.cuh"

#include "debug.h"
#include "vec3.cuh"
#include "sphere.cuh"
#include "scene.cuh"

#include <fstream>
#include <string>

namespace importer
{

__global__ void add_sphere_to_scene(Scene * scene, vec3 color, vec3 position, float radius)
{
	scene->add(new Sphere{ color, position, radius });
}

__global__ void set_scene_camera(Scene * scene, vec3 position, vec3 right, vec3 up, vec3 center)
{
	scene->set_camera(Camera{ position, right, up, center });
}

void import_scene(const char * path, Scene * scene)
{
	std::ifstream file{ path };
	if (file.is_open())
	{
		// Parse
		std::string line;
		while (std::getline(file, line))
		{
			switch (line[0])
			{
			case 'C':
			{
				vec3 position, right, up, center;
				
				// Projection center, right, up, distance to projection center
				sscanf_s(line.c_str(), "C (%f,%f,%f) (%f,%f,%f) (%f,%f,%f) %f",
					&center.x, &center.y, &center.z,
					&right.x, &right.y, &right.z, 
					&up.x, &up.y, &up.z, 
					&position.x);

				// Compute position
				position = center + vec3::normalize(vec3::cross(right, up)) * position.x;

				// Upload to GPU
				set_scene_camera <<<1,1>>>(scene, position, right, up, center);
				CheckCUDAError(hipGetLastError());
				CheckCUDAError(hipDeviceSynchronize());
				break;
			}
			case 'S':
			{
				vec3 position, color;
				float radius;

				// Position and radius
				sscanf_s(line.c_str(), "S (%f,%f,%f) %f",
					&position.x, &position.y, &position.z, &radius);

				// Material
				std::getline(file, line);
				sscanf_s(line.c_str(), "(%f,%f,%f)", &color.r, &color.g, &color.b);

				// Upload to GPU
				add_sphere_to_scene<<<1,1>>>(scene, color, position, radius);
				CheckCUDAError(hipGetLastError());
				CheckCUDAError(hipDeviceSynchronize());
				break;
			}
			default:
				break;
			}
		}
	}
}

}
