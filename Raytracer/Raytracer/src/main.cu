#include "hip/hip_runtime.h"
#include "debug.h"
#include "vec3.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "scene.cuh"
#include "importer.cuh"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#include "stb/stb_image_write.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

const int width = 500;
const int height = 500;
const int buffer_size = width * height * 3;
const int half_width = width / 2;
const int half_height = height / 2;

__device__ vec3 cast_ray(float x, float y, Scene * scene)
{
	// Compute ray
	float current_x = (static_cast<float>(x) + 0.5f - half_width) / half_width;
	float current_y = -(static_cast<float>(y) + 0.5f - half_height) / half_height;
	const Camera & camera = scene->camera();
	vec3 pixel_position = camera.projection_center() + camera.right() * current_x + camera.up() * current_y;
	Ray ray{ camera.position(), vec3::normalize(pixel_position - camera.position()) };

	// Compute color
	vec3 final_color{ 0.0f };
	CollisionData collision_data;
	const vector<Surface *> & surfaces = scene->surfaces();
	for (int i = 0; i < surfaces.size(); ++i)
	{
		if (surfaces[i]->collide(ray, 0.0f, collision_data.mT, collision_data))
			final_color += collision_data.mColor;
	}
	return final_color;
}

__global__ void render_image(unsigned char * image_data, int width, int height, Scene * scene)
{
	// Get coordinates from block and thread indices
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= width || y >= height) return;
	int pixel_index = y * width * 3 + x * 3;

	// Compute and store color
	vec3 color = cast_ray(static_cast<float>(x), static_cast<float>(y), scene);

	image_data[pixel_index] = static_cast<unsigned char>(color.r * 255.99f);
	image_data[pixel_index + 1] = static_cast<unsigned char>(color.g * 255.99f);
	image_data[pixel_index + 2] = static_cast<unsigned char>(color.b * 255.99f);
}

__global__ void initialize_scene(Scene * scene)
{
	new (scene) Scene{};
}

__global__ void populate_scene(Scene * scene)
{
	vec3 color{ 1.0f, 0.0f, 0.0f };
	vec3 center{ 0.0f, 0.0f, -5.0f };
	float radius = 0.5f;
	scene->add(new Sphere{ color, center, radius });
	color = vec3{ 0.0f, 1.0f, 0.0f };
	center.x = 1.0f;
	scene->add(new Sphere{ color, center, radius });
	color = vec3{ 0.0f, 0.0f, 1.0f };
	center.y = 1.0f;
	scene->add(new Sphere{ color, center, radius });
	color = vec3{ 1.0f, 1.0f, 0.0f };
	center.x = 0.0f;
	scene->add(new Sphere{ color, center, radius });
}

__global__ void destroy_scene(Scene * scene)
{
	scene->~Scene();
}

int main()
{
	// Scene creation
	Scene * scene = nullptr;
	CheckCUDAError(hipMalloc((void **)&scene, sizeof(Scene)));
	initialize_scene<<<1,1>>>(scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());
	importer::import_scene("scene.txt", scene);

	// Allocate memory in GPU
	unsigned char * image_data = nullptr;
	CheckCUDAError(hipMallocManaged((void **)&image_data, buffer_size));

	// Compute needed blocks for the whole image
	dim3 threads(8, 8);
	dim3 blocks(width / threads.x + 1, height / threads.y + 1);

	// Render image
	render_image<<<blocks,threads>>>(image_data, width, height, scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());

	// Store color
	stbi_write_png("MyOutput.png", width, height, 3, image_data, 0);

	// Free memory
	hipFree(image_data);
	destroy_scene<<<1,1>>>(scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());
	hipFree(scene);
	return 0;
}
