#include "hip/hip_runtime.h"
#include "debug.h"
#include "ray.cuh"
#include "sphere.cuh"
#include "scene.cuh"
#include "importer.cuh"

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#define STB_IMAGE_WRITE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#include <stb/stb_image_write.h>

#include <hip/hip_runtime.h>
#include <>

#include <iostream>

const int width = 500;
const int height = 500;
const int buffer_size = width * height * 3;
const int half_width = width / 2;
const int half_height = height / 2;

__device__ glm::vec3 cast_ray(float x, float y, Scene * scene)
{
	// Compute ray
	float current_x = (static_cast<float>(x) + 0.5f - half_width) / half_width;
	float current_y = -(static_cast<float>(y) + 0.5f - half_height) / half_height;
	const Camera & camera = scene->camera();
	glm::vec3 pixel_position = camera.projection_center() + camera.right() * current_x + camera.up() * current_y;
	Ray ray{ camera.position(), glm::normalize(pixel_position - camera.position()) };
	glm::vec3 final_color{ 0.0f };
	float attenuation_inverse = 1.0f;

	for (int ray_count = 0; ray_count < 10; ++ray_count)
	{
		// Check for collisions
		CollisionData collision_data;
		const vector<Surface *> & surfaces = scene->surfaces();
		for (int i = 0; i < surfaces.size(); ++i)
			surfaces[i]->collide(ray, 0.0f, collision_data.mT, collision_data);

		// Hit nothing, exit
		if (collision_data.mT == FLT_MAX)
			break;

		// Shadow check
		final_color += collision_data.mMaterial.mColor * scene->ambient() * attenuation_inverse;
		glm::vec3 collision_point = ray.at(collision_data.mT);
		glm::vec3 reflected = glm::reflect(ray.direction(), collision_data.mNormal);
		const vector<PointLight> & lights = scene->lights();
		for (int i = 0; i < lights.size(); ++i)
		{
			CollisionData dummy;
			ray = Ray{ collision_point, lights[i].position() - collision_point };
			int shadow_count = 0;
			
			// Check for collisions, if there's any, we are in shadow
			for(int j = 0; j < surfaces.size(); ++j)
			{
				if (surfaces[j]->collide(ray, 0.001f, 1.0f, dummy))
					shadow_count++;
			}
			
			// In shadow
			if (shadow_count)
				continue;
			
			// Diffuse
			glm::vec3 to_light = glm::normalize(ray.direction());
			float cos_angle = glm::dot(to_light, collision_data.mNormal);
			if (cos_angle < 0.0f)
				continue;
			final_color += collision_data.mMaterial.mColor * lights[i].intensity() * cos_angle * attenuation_inverse;

			// Specular
			cos_angle = glm::dot(reflected, to_light);
			if (cos_angle > 0.0f)
				final_color += lights[i].intensity() * powf(cos_angle, collision_data.mMaterial.mShininess) * collision_data.mMaterial.mSpecularCoefficient * attenuation_inverse;

		}

		if (collision_data.mMaterial.mSpecularCoefficient)
		{
			attenuation_inverse *= collision_data.mMaterial.mSpecularCoefficient;
			ray = Ray{ collision_point + collision_data.mNormal * 0.001f, reflected };
		}
		else break;
	}

	return glm::min(final_color, glm::vec3{ 1.0f });
}

__global__ void render_image(unsigned char * image_data, int width, int height, Scene * scene)
{
	// Get coordinates from block and thread indices
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x >= width || y >= height) return;
	int pixel_index = y * width * 3 + x * 3;

	// Compute and store color
	glm::vec3 color = cast_ray(static_cast<float>(x), static_cast<float>(y), scene);

	image_data[pixel_index] = static_cast<unsigned char>(color.r * 255.99f);
	image_data[pixel_index + 1] = static_cast<unsigned char>(color.g * 255.99f);
	image_data[pixel_index + 2] = static_cast<unsigned char>(color.b * 255.99f);
}

__global__ void initialize_scene(Scene * scene)
{
	new (scene) Scene{};
}

__global__ void destroy_scene(Scene * scene)
{
	scene->~Scene();
}

int main()
{
	// Scene creation
	Scene * scene = nullptr;
	CheckCUDAError(hipMalloc((void **)&scene, sizeof(Scene)));
	initialize_scene<<<1,1>>>(scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());
	importer::import_scene("scene.txt", scene);

	// Allocate memory in GPU
	unsigned char * image_data = nullptr;
	CheckCUDAError(hipMallocManaged((void **)&image_data, buffer_size));

	// Compute needed blocks for the whole image
	dim3 threads(8, 8);
	dim3 blocks(width / threads.x + 1, height / threads.y + 1);

	// Render image
	render_image<<<blocks,threads>>>(image_data, width, height, scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());

	// Store color
	stbi_write_png("MyOutput.png", width, height, 3, image_data, 0);

	// Free memory
	hipFree(image_data);
	destroy_scene<<<1,1>>>(scene);
	CheckCUDAError(hipGetLastError());
	CheckCUDAError(hipDeviceSynchronize());
	hipFree(scene);
	return 0;
}
